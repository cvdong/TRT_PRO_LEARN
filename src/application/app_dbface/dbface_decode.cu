#include "hip/hip_runtime.h"
#include <common/cuda_tools.hpp>
#include <common/ilogger.hpp>
#include <stdio.h>
#include <iostream>

using namespace std;

namespace DBFace{

    static const int NUM_BOX_ELEMENT = 17; // left, top, right, bottom, confidence, class, keepflag, 10landmark

    static __device__ float common_exp(float value) {

		float gate = 1;
		float base = exp(gate);
		if (fabs(value) < gate)
			return value * base;

		if (value > 0) {
			return exp(value);
		}
		else {
			return -exp(-value);
		}
	}

    static __host__ float desigmoid(float y){
        return -log(1.0f / y - 1.0f);
    }

    static __device__ float sigmoid(float x){
        return 1.0f / (1.0f + exp(-x));
    }

    static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
        *ox = matrix[0] * x + matrix[1] * y + matrix[2];
        *oy = matrix[3] * x + matrix[4] * y + matrix[5];
    }

    static __global__ void decode_kernel(
        float* pool_hm_ptr, float* hm_ptr, float* tlrb_ptr, float* landmark_ptr, int num_bboxes,
        int fm_width, int fm_height, int stride,
        float conf_T, float* invert_affine_matrix, float* parray, int max_objects
    ){
        /* 
        On the right, we access a thread in GPU by only using blockIdx.x, blockDim.x and threadIdx.x for 
        simplicity, which indicates which block ,firstly, the gpu thread is in and within the given block, which thread the gpu thread
        is. In other words, we actually do an ndim-indexes-to-1-dim-index conversion for indexing a gpu thread.
         */
        int position = blockIdx.x * blockDim.x + threadIdx.x;
        if (position >= num_bboxes)
            return;

        float confidence = hm_ptr[position];
        if(pool_hm_ptr && confidence != pool_hm_ptr[position]) // when the network is DBFaceSmall, pool_hm_ptr is nullptr
            return;

        if(confidence < conf_T)
            return;
        
        /*
        pitem_ptr is the addr of anyone of the pixels on the 128 x 128 2d feature map in python but
        in nature is a 16384 feature 1d array in c++.
        Along the pixel, we can access its relavant (2+2+80+80) infomation as follows:
            2: regxy
            2: wh
            80: hm
            80: pooled_hm
        
        Now (2+2+80+80) channels need to be anaylized to compute the bboxes.
        Steps can be summaried as the follows:
            - 1. for loop the hm and pool_hm channels to get the class with largest confidence (aka argmax).
                After that, a mask indicating the obj (y,x) location is produced, which is named as obj_kept_mask.
            - 2. During the for loop mentioned above, we add the x,y with regx and regy, which is followed using w and h to compute the x,y,r,b
                on featuremap. Finally, we do the inverse-downsampling(x4) to restore the real x,y,r,b.  
   
         */
        // detect an object. parray should be added by 1.
        int index = atomicAdd(parray, 1);
        if (index >= max_objects) return;

        float cx = position % fm_width;
        float cy = position / fm_height;
        float left   = (cx - tlrb_ptr[num_bboxes * 0 + position]) * stride; // (y_,x_) is the center of an obj on the featuremap scale.
        float top    = (cy - tlrb_ptr[num_bboxes * 1 + position]) * stride; // the sybol here with the trailing underscore refers to featuremap scale.
        float right  = (cx + tlrb_ptr[num_bboxes * 2 + position]) * stride;
        float bottom = (cy + tlrb_ptr[num_bboxes * 3 + position]) * stride;

        affine_project(invert_affine_matrix, left,  top,    &left,  &top); // modify the xyrb in place.
        affine_project(invert_affine_matrix, right, bottom, &right, &bottom);
        
        float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
        *pout_item++ = left;
        *pout_item++ = top;
        *pout_item++ = right;
        *pout_item++ = bottom;
        *pout_item++ = confidence;
        *pout_item++ = 0;
        *pout_item++ = 1; // keep = 1  ignore = 0

        for(int i = 0; i < 5; ++i){
            float x = landmark_ptr[num_bboxes * i + position] * 4;
            float y = landmark_ptr[num_bboxes * (5 + i) + position] * 4;
            x = (common_exp(x) + cx) * stride;
            y = (common_exp(y) + cy) * stride;
            affine_project(invert_affine_matrix, x, y, &x, &y);
            *pout_item++ = x;
            *pout_item++ = y;
        }
    };

    static __device__ float box_iou(
        float aleft, float atop, float aright, float abottom, 
        float bleft, float btop, float bright, float bbottom
    ){

        float cleft 	= max(aleft, bleft);
        float ctop 		= max(atop, btop);
        float cright 	= min(aright, bright);
        float cbottom 	= min(abottom, bbottom);
        
        float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
        if(c_area == 0.0f)
            return 0.0f;
        
        float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
        float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
        return c_area / (a_area + b_area - c_area);
    }

    static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold){
        // refer to tensorRT_cpp/tutorial/2.0CenterNet_from_torch_trt/nms_cuda.jpg and comments.jpg for understanding.
        int position = (blockDim.x * blockIdx.x + threadIdx.x);
        int count = min((int)*bboxes, max_objects);
        if (position >= count) 
            return;
        
        // left, top, right, bottom, confidence, class, keepflag
        float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
        for(int i = 0; i < count; ++i){
            float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
            if(i == position || pcurrent[5] != pitem[5]) continue;

            if(pitem[4] >= pcurrent[4]){
                if(pitem[4] == pcurrent[4] && i < position)
                    continue;

                float iou = box_iou(
                    pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                    pitem[0],    pitem[1],    pitem[2],    pitem[3]
                );

                if(iou > threshold){
                    pcurrent[6] = 0;  // 1=keep, 0=ignore
                    return;
                }
            }
        }
    } 

    void decode_kernel_invoker(float* pool_hm_ptr, float* hm_ptr, float* tlrb_ptr, float* landmark_ptr,
                int fm_width, int fm_height, int stride,
                float conf_T, float nms_threshold, float* invert_affine_matrix, 
                float* parray, int max_objects, hipStream_t stream){
        
        int num_bboxes = fm_width * fm_height;
        auto grid = CUDATools::grid_dims(num_bboxes);
        auto block = CUDATools::block_dims(num_bboxes);

        checkCudaKernel(decode_kernel<<<grid, block, 0, stream>>>(pool_hm_ptr, hm_ptr, tlrb_ptr, landmark_ptr,
            num_bboxes, fm_width, fm_height, stride, conf_T,
            invert_affine_matrix, parray, max_objects));

        grid = CUDATools::grid_dims(max_objects);
        block = CUDATools::block_dims(max_objects);
        checkCudaKernel(nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
    };
} // namespace Centernet
